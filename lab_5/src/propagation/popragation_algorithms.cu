#include "hip/hip_runtime.h"
#include "popragation_algorithms.hpp"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <cmath>

namespace py = pybind11;

/**
 * @brief Sigmoid activation function
 * @param x Input value
 * @return Sigmoid of x
 */
__host__ __device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));  // Use expf instead of exp for GPU compatibility
}

/**
 * @brief Sigmoid derivative function
 * @param x Input value
 * @return Derivative of sigmoid at x
 */
__host__ __device__ float sigmoid_derivative(float x) {
    return x * (1.0f - x);
}

/**
 * @brief ReLU activation function
 * @param x Input value
 * @return ReLU of x
 */
__host__ __device__ float relu(float x) {
    return x > 0.0f ? x : 0.0f;
}

/**
 * @brief ReLU derivative function
 * @param x Input value
 * @return Derivative of ReLU at x
 */
__host__ __device__ float relu_derivative(float x) {
    return x > 0.0f ? 1.0f : 0.0f;
}


/**
 * @brief Forward propagation through a simple 2-layer neural network
 * @param x Input vector (2 elements)
 * @param W1 Weights for the first layer (2x4)
 * @param b1 Biases for the first layer (4)
 * @param W2 Weights for the second layer (4x1)
 * @param b2 Biases for the second layer (1)
 * @param z1 Output of the first layer before activation (4)
 * @param a1 Output of the first layer after activation (4)
 * @param z2 Output of the second layer before activation (1)
 * @param a2 Output of the second layer after activation (1)
 * @param use_relu Whether to use ReLU activation in the hidden layer
 */
__host__ __device__
void forward_propagation(
    const float* x,                     // input 2-elementowy
    const float* W1, const float* b1,   // W1: 2x4, b1: 4
    const float* W2, const float* b2,   // W2: 4x1, b2: 1
    float* z1, float* a1,               // z1: 4, a1: 4
    float& z2, float& a2,               // z2, a2: 1
    bool use_relu = false               // wybór aktywacji
) {
    // Warstwa ukryta
    for (int i = 0; i < 4; ++i) {
        z1[i] = b1[i];
        for (int j = 0; j < 2; ++j) {
            z1[i] += x[j] * W1[j * 4 + i];  // W1: [2][4]
        }
        a1[i] = use_relu ? relu(z1[i]) : sigmoid(z1[i]);
    }

    // Warstwa wyjściowa
    z2 = b2[0];
    for (int i = 0; i < 4; ++i) {
        z2 += a1[i] * W2[i];  // W2: [4][1]
    }
    a2 = sigmoid(z2);  // Na wyjściu zawsze sigmoid
}

/**
 * @brief Backward propagation to compute gradients for weights and biases
 * @param x Input vector (2 elements)
 * @param y Target output (1 element)
 * @param W1 Weights for the first layer (2x4)
 * @param W2 Weights for the second layer (4x1)
 * @param z1 Output of the first layer before activation (4)
 * @param a1 Output of the first layer after activation (4)
 * @param z2 Output of the second layer before activation (1)
 * @param a2 Output of the second layer after activation (1)
 * @param dW1 Gradient for W1 (2x4)
 * @param db1 Gradient for b1 (4)
 * @param dW2 Gradient for W2 (4x1)
 * @param db2 Gradient for b2 (1)
 * @param use_relu Whether to use ReLU activation in the hidden layer
 */
__host__ __device__
void backward_propagation(
    const float* x,                      // input 2-elementowy
    const float* y,                      // target (1-elementowy)
    const float* W1, const float* W2,    // wagi
    const float* z1, const float* a1,    // z i a z forwarda
    float z2, float a2,                  // z2 i a2 z forwarda
    float* dW1, float* db1,              // gradienty
    float* dW2, float* db2,
    bool use_relu = false
) {
    // Warstwa wyjściowa
    float dz2 = a2 - y[0];
    for (int i = 0; i < 4; ++i) {
        dW2[i] = a1[i] * dz2;
    }
    db2[0] = dz2;

    // Warstwa ukryta
    for (int i = 0; i < 4; ++i) {
        float da1 = W2[i] * dz2;
        float dz1 = da1 * (use_relu ? relu_derivative(z1[i]) : sigmoid_derivative(a1[i]));

        db1[i] = dz1;

        for (int j = 0; j < 2; ++j) {
            dW1[j * 4 + i] = x[j] * dz1;  // W1: [2][4]
        }
    }
}

// Pybind11 module definition
PYBIND11_MODULE(cuda_nn, m) {
    m.doc() = "CUDA Neural Network Propagation Module";
    
    // Expose activation functions
    m.def("sigmoid", &sigmoid, "Sigmoid activation function");
    m.def("sigmoid_derivative", &sigmoid_derivative, "Sigmoid derivative");
    m.def("relu", &relu, "ReLU activation function");
    m.def("relu_derivative", &relu_derivative, "ReLU derivative");
    
    // Expose main functions (you'll need to implement propagate and predict)
    // m.def("propagate", &propagate, "Forward and backward propagation");
    // m.def("predict", &predict, "Prediction function");
}
