#include "hip/hip_runtime.h"
#include "popragation_algorithms.hpp"
#include "propagation_algorithms_gpu.hpp"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>

namespace py = pybind11;

/**
 * @brief CUDA kernel for forward propagation
 */
__global__ void forward_propagation_kernel(
    const float* x,                     // input 2-elementowy
    const float* W1, const float* b1,   // W1: 2x4, b1: 4
    const float* W2, const float* b2,   // W2: 4x1, b2: 1
    float* z1, float* a1,               // z1: 4, a1: 4
    float* z2, float* a2,               // z2: 1, a2: 1
    bool use_relu                       // wybór aktywacji
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Warstwa ukryta - każdy wątek oblicza jeden neuron
    if (idx < 4) {
        z1[idx] = b1[idx];
        for (int j = 0; j < 2; ++j) {
            z1[idx] += x[j] * W1[j * 4 + idx];  // W1: [2][4]
        }
        a1[idx] = use_relu ? relu(z1[idx]) : sigmoid(z1[idx]);
    }
    
    // Synchronizacja przed obliczeniem warstwy wyjściowej
    __syncthreads();
    
    // Warstwa wyjściowa - tylko pierwszy wątek
    if (idx == 0) {
        *z2 = b2[0];
        for (int i = 0; i < 4; ++i) {
            *z2 += a1[i] * W2[i];  // W2: [4][1]
        }
        *a2 = sigmoid(*z2);  // Na wyjściu zawsze sigmoid
    }
}

/**
 * @brief CUDA kernel for backward propagation
 */
__global__ void backward_propagation_kernel(
    const float* x,                      // input 2-elementowy
    const float* y,                      // target (1-elementowy)
    const float* W1, const float* W2,    // wagi
    const float* z1, const float* a1,    // z i a z forwarda
    float z2, float a2,                  // z2 i a2 z forwarda
    float* dW1, float* db1,              // gradienty
    float* dW2, float* db2,
    bool use_relu
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Warstwa wyjściowa - obliczana przez pierwszy wątek
    float dz2;
    if (idx == 0) {
        dz2 = a2 - y[0];
        for (int i = 0; i < 4; ++i) {
            dW2[i] = a1[i] * dz2;
        }
        db2[0] = dz2;
    }
    
    // Synchronizacja aby mieć dz2
    __syncthreads();
    dz2 = a2 - y[0];  // Ponowne obliczenie dla wszystkich wątków
    
    // Warstwa ukryta - każdy wątek oblicza gradienty dla jednego neuronu
    if (idx < 4) {
        float da1 = W2[idx] * dz2;
        float dz1 = da1 * (use_relu ? relu_derivative(z1[idx]) : sigmoid_derivative(a1[idx]));

        db1[idx] = dz1;

        for (int j = 0; j < 2; ++j) {
            dW1[j * 4 + idx] = x[j] * dz1;  // W1: [2][4]
        }
    }
}

// GPU-accelerated wrapper functions
py::array_t<float> forward_propagation_gpu(
    py::array_t<float> x_input,
    py::array_t<float> W1_input,
    py::array_t<float> b1_input,
    py::array_t<float> W2_input,
    py::array_t<float> b2_input,
    bool use_relu = false
) {
    auto x_buf = x_input.request();
    auto W1_buf = W1_input.request();
    auto b1_buf = b1_input.request();
    auto W2_buf = W2_input.request();
    auto b2_buf = b2_input.request();
    
    // Allocate GPU memory
    float *d_x, *d_W1, *d_b1, *d_W2, *d_b2;
    float *d_z1, *d_a1, *d_z2, *d_a2;
    
    hipMalloc(&d_x, 2 * sizeof(float));
    hipMalloc(&d_W1, 8 * sizeof(float));  // 2x4
    hipMalloc(&d_b1, 4 * sizeof(float));
    hipMalloc(&d_W2, 4 * sizeof(float));
    hipMalloc(&d_b2, 1 * sizeof(float));
    hipMalloc(&d_z1, 4 * sizeof(float));
    hipMalloc(&d_a1, 4 * sizeof(float));
    hipMalloc(&d_z2, 1 * sizeof(float));
    hipMalloc(&d_a2, 1 * sizeof(float));
    
    // Copy data to GPU
    hipMemcpy(d_x, x_buf.ptr, 2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W1, W1_buf.ptr, 8 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, b1_buf.ptr, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, W2_buf.ptr, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, b2_buf.ptr, 1 * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 blockSize(4);
    dim3 gridSize(1);
    forward_propagation_kernel<<<gridSize, blockSize>>>(
        d_x, d_W1, d_b1, d_W2, d_b2, d_z1, d_a1, d_z2, d_a2, use_relu
    );
    
    // Wait for kernel to complete
    hipDeviceSynchronize();
    
    // Copy results back to host
    float h_a1[4], h_z2, h_a2;
    hipMemcpy(h_a1, d_a1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_z2, d_z2, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_a2, d_a2, 1 * sizeof(float), hipMemcpyDeviceToHost);
    
    // Create result array
    auto result = py::array_t<float>(6);
    auto result_buf = result.request();
    float* result_ptr = static_cast<float*>(result_buf.ptr);
    
    for (int i = 0; i < 4; i++) {
        result_ptr[i] = h_a1[i];
    }
    result_ptr[4] = h_z2;
    result_ptr[5] = h_a2;
    
    // Free GPU memory
    hipFree(d_x); hipFree(d_W1); hipFree(d_b1); hipFree(d_W2); hipFree(d_b2);
    hipFree(d_z1); hipFree(d_a1); hipFree(d_z2); hipFree(d_a2);
    
    return result;
}

py::dict backward_propagation_gpu(
    py::array_t<float> x_input,
    py::array_t<float> y_input,
    py::array_t<float> W1_input,
    py::array_t<float> W2_input,
    py::array_t<float> z1_input,
    py::array_t<float> a1_input,
    float z2,
    float a2,
    bool use_relu = false
) {
    auto x_buf = x_input.request();
    auto y_buf = y_input.request();
    auto W1_buf = W1_input.request();
    auto W2_buf = W2_input.request();
    auto z1_buf = z1_input.request();
    auto a1_buf = a1_input.request();
    
    // Allocate GPU memory
    float *d_x, *d_y, *d_W1, *d_W2, *d_z1, *d_a1;
    float *d_dW1, *d_db1, *d_dW2, *d_db2;
    
    hipMalloc(&d_x, 2 * sizeof(float));
    hipMalloc(&d_y, 1 * sizeof(float));
    hipMalloc(&d_W1, 8 * sizeof(float));
    hipMalloc(&d_W2, 4 * sizeof(float));
    hipMalloc(&d_z1, 4 * sizeof(float));
    hipMalloc(&d_a1, 4 * sizeof(float));
    hipMalloc(&d_dW1, 8 * sizeof(float));
    hipMalloc(&d_db1, 4 * sizeof(float));
    hipMalloc(&d_dW2, 4 * sizeof(float));
    hipMalloc(&d_db2, 1 * sizeof(float));
    
    // Copy data to GPU
    hipMemcpy(d_x, x_buf.ptr, 2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y_buf.ptr, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W1, W1_buf.ptr, 8 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, W2_buf.ptr, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z1, z1_buf.ptr, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_a1, a1_buf.ptr, 4 * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 blockSize(4);
    dim3 gridSize(1);
    backward_propagation_kernel<<<gridSize, blockSize>>>(
        d_x, d_y, d_W1, d_W2, d_z1, d_a1, z2, a2, d_dW1, d_db1, d_dW2, d_db2, use_relu
    );
    
    // Wait for kernel to complete
    hipDeviceSynchronize();
    
    // Copy results back to host
    float h_dW1[8], h_db1[4], h_dW2[4], h_db2[1];
    hipMemcpy(h_dW1, d_dW1, 8 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_db1, d_db1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_dW2, d_dW2, 4 * sizeof(float), hipMemcpyDeviceToHost);  
    hipMemcpy(h_db2, d_db2, 1 * sizeof(float), hipMemcpyDeviceToHost);
    
    // Create numpy arrays for gradients
    auto dW1_array = py::array_t<float>({2, 4});
    auto db1_array = py::array_t<float>(4);
    auto dW2_array = py::array_t<float>(4);
    auto db2_array = py::array_t<float>(1);
    
    auto dW1_buf = dW1_array.request();
    auto db1_buf = db1_array.request();
    auto dW2_buf = dW2_array.request();
    auto db2_buf = db2_array.request();
    
    float* dW1_ptr = static_cast<float*>(dW1_buf.ptr);
    float* db1_ptr = static_cast<float*>(db1_buf.ptr);
    float* dW2_ptr = static_cast<float*>(dW2_buf.ptr);
    float* db2_ptr = static_cast<float*>(db2_buf.ptr);
    
    // Copy gradients
    for (int i = 0; i < 8; i++) dW1_ptr[i] = h_dW1[i];
    for (int i = 0; i < 4; i++) db1_ptr[i] = h_db1[i];
    for (int i = 0; i < 4; i++) dW2_ptr[i] = h_dW2[i];
    db2_ptr[0] = h_db2[0];
    
    // Free GPU memory
    hipFree(d_x); hipFree(d_y); hipFree(d_W1); hipFree(d_W2);
    hipFree(d_z1); hipFree(d_a1); hipFree(d_dW1); hipFree(d_db1);
    hipFree(d_dW2); hipFree(d_db2);
    
    py::dict gradients;
    gradients["dW1"] = dW1_array;
    gradients["db1"] = db1_array;
    gradients["dW2"] = dW2_array;
    gradients["db2"] = db2_array;
    
    return gradients;
}
